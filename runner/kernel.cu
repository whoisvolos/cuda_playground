#include "hip/hip_runtime.h"
/*
* This program uses the device CURAND API to calculate what
* proportion of pseudo-random ints have low bit set.
*/
#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_math_constants.h>
#include <>
#include "math.cuh"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void random_init(hiprandState_t* states, unsigned long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
};

__global__ void generate_random(hiprandState_t* statesRad, hiprandState_t* statesPhi, float3* results, const int samples) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t stateRad = statesRad[idx];
    hiprandState_t statePhi = statesPhi[idx];
    while (idx < samples) {
        float r = hiprand_uniform(&stateRad);
        float phi = hiprand_uniform(&statePhi) * 2 * HIP_PI_F;
        float rad = sqrtf(r);
        results[idx] = make_float3(rad * cosf(phi), rad * sinf(phi), sqrtf(1 - r));
        idx += blockDim.x * gridDim.x;
    }
};

int main(int argc, char *argv[]) {
    atexit([] { _getch(); });

    int devCount;
    hipGetDeviceCount(&devCount);
    for (int i = 0; i < devCount; ++i) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("%i: %s (compat %i.%i)\n", i, props.name, props.major, props.minor);
    }

    hiprandState_t* statesRad;
    hiprandState_t* statesPhi;
    int BLOCKS = 256;
    int TPB = 512;
    const int samples = BLOCKS * TPB * 256;
    printf("Trials: %i, %i Mb\n", samples, (samples * sizeof(float3) + BLOCKS * TPB * 2 * sizeof(hiprandState_t)) / 1024 / 1024);
    int TRIALS = 64;
    StopWatchInterface *hTimer;

    hipSetDevice(0);
    hipSetDeviceFlags(hipDeviceScheduleSpin);

    checkCudaErrors(hipMalloc((void **)&statesRad, BLOCKS * TPB * sizeof(hiprandState_t)));
    checkCudaErrors(hipMalloc((void **)&statesPhi, BLOCKS * TPB * sizeof(hiprandState_t)));

    printf("Initializing random\n");
    random_init << <BLOCKS, TPB >> >(statesRad, 0l);
    random_init << <BLOCKS, TPB >> >(statesPhi, 1234l);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipPeekAtLastError());
    printf("Initializing random done\n");

    float3* devRaysDirection;
    checkCudaErrors(hipMalloc((void **)&devRaysDirection, (size_t)samples * sizeof(float3)));

    sdkCreateTimer(&hTimer);
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for (int i = 0; i < TRIALS; ++i) {
        generate_random << <BLOCKS, TPB >> >(statesRad, statesPhi, devRaysDirection, samples);
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipPeekAtLastError());
        printf("Done trial %i\n", i);
    }
    sdkStopTimer(&hTimer);

    printf("%f Grays/s\n", (float)TRIALS * samples * 1e-9 * 1e+3 / sdkGetTimerValue(&hTimer));

    //float3* rays = new float3[samples];
    //CUDA_CALL(hipMemcpy(rays, devRaysDirection, sizeof(float3) * samples, hipMemcpyDeviceToHost));
    //delete rays;
    
    hipFree(devRaysDirection);

    hipFree(statesRad);
    hipFree(statesPhi);

    return EXIT_SUCCESS;
}