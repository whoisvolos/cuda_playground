#include "hip/hip_runtime.h"
/*
* This program uses the device CURAND API to calculate what
* proportion of pseudo-random ints have low bit set.
*/
#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_math_constants.h>
#include <>
#include "math.cuh"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void random_init(hiprandState_t* states, unsigned long long seedRad, unsigned long long seedPhi) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x * 2;
    hiprand_init(seedRad, idx, 0, &states[idx]);
    hiprand_init(seedPhi, idx + 1, 0, &states[idx + 1]);
};

__global__ void generate_random(hiprandState_t* states, float3* results, const int samples) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stateIdx = idx * 2;
    while (idx < samples) {
        float r = hiprand_uniform(&states[stateIdx]);
        float phi = hiprand_uniform(&states[stateIdx + 1]) * 2 * HIP_PI_F;
        float rad = sqrtf(r);
        results[idx] = make_float3(rad * cosf(phi), rad * sinf(phi), sqrtf(1 - r));
        idx += blockDim.x * gridDim.x;
    }
};

int main(int argc, char *argv[]) {
    atexit([] { _getch(); });

    int devCount;
    hipGetDeviceCount(&devCount);
    for (int i = 0; i < devCount; ++i) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("%i: %s (compat %i.%i)\n", i, props.name, props.major, props.minor);
    }

    hiprandState_t* states;
    int BLOCKS = 256;
    int TPB = 512;
    const int samples = 134217728;// 33554432; // 32M rays
    int TRIALS = 1000;
    StopWatchInterface *hTimer;

    hipSetDevice(0);
    hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

    CUDA_CALL(hipMalloc((void **)&states, sizeof(BLOCKS * TPB * sizeof(hiprandState_t) * 2)));

    random_init << <BLOCKS, TPB >> >(states, 0l, 1234l);
    checkCudaErrors(hipPeekAtLastError());

    float3* devRaysDirection;
    CUDA_CALL(hipMalloc((void **)&devRaysDirection, (size_t)samples * sizeof(float3)));

    sdkCreateTimer(&hTimer);
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for (int i = 0; i < TRIALS; ++i) {
        generate_random << <BLOCKS, TPB >> >(states, devRaysDirection, samples);
    }
    sdkStopTimer(&hTimer);
    checkCudaErrors(hipPeekAtLastError());
    //checkCudaErrors(hipDeviceSynchronize());

    printf("%f Gigarays/s\n", (float)TRIALS * samples * 1e-9 / sdkGetTimerValue(&hTimer));

    //float3* rays = new float3[samples];
    //CUDA_CALL(hipMemcpy(rays, devRaysDirection, sizeof(float3) * samples, hipMemcpyDeviceToHost));
    //delete rays;
    
    hipFree(devRaysDirection);

    hipFree(states);

    return EXIT_SUCCESS;
}