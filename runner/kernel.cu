#include "hip/hip_runtime.h"
/*
* This program uses the device CURAND API to calculate what
* proportion of pseudo-random ints have low bit set.
*/
#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hip/hip_math_constants.h>
#include <>
#include "math.cuh"

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void random_init(hiprandState_t* states, unsigned long long seedRad, unsigned long long seedPhi) {
    int idx = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    hiprand_init(seedRad, idx, 0, &states[idx]);
    hiprand_init(seedPhi, idx + 1, 0, &states[idx + 1]);
};

__global__ void generate_random(hiprandState_t* states, float3* results, const int samples) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t stateRad = states[idx * 2];
    hiprandState_t statePhi = states[idx * 2 + 1];
    while (idx < samples) {
        float r = hiprand_uniform(&stateRad);
        float phi = hiprand_uniform(&statePhi) * 2 * HIP_PI_F;
        float rad = sqrtf(r);
        results[idx] = make_float3(rad * cosf(phi), rad * sinf(phi), sqrtf(1 - r));
        idx += blockDim.x * gridDim.x;
    }
};

int main(int argc, char *argv[]) {
    atexit([] { _getch(); });

    int devCount;
    hipGetDeviceCount(&devCount);
    for (int i = 0; i < devCount; ++i) {
        hipDeviceProp_t props;
        hipGetDeviceProperties(&props, i);
        printf("%i: %s (compat %i.%i)\n", i, props.name, props.major, props.minor);
    }

    hiprandState_t* states;
    int BLOCKS = 256;
    int TPB = 512;
    const int samples = BLOCKS * TPB * 256;
    printf("Trials: %i, %i Mb\n", samples, (samples * sizeof(float3) + BLOCKS * TPB * 2 * sizeof(hiprandState_t)) / 1024 / 1024);
    int TRIALS = 10;
    StopWatchInterface *hTimer;

    hipSetDevice(0);
    hipSetDeviceFlags(hipDeviceScheduleSpin);

    checkCudaErrors(hipMalloc((void **)&states, BLOCKS * TPB * sizeof(hiprandState_t) * 2));

    printf("Initializing random\n");
    random_init << <BLOCKS, TPB >> >(states, 0l, 1234l);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipPeekAtLastError());
    printf("Initializing random done\n");

    float3* devRaysDirection;
    checkCudaErrors(hipMalloc((void **)&devRaysDirection, (size_t)samples * sizeof(float3)));

    sdkCreateTimer(&hTimer);
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);

    for (int i = 0; i < TRIALS; ++i) {
        generate_random << <BLOCKS, TPB >> >(states, devRaysDirection, samples);
        checkCudaErrors(hipDeviceSynchronize());
        checkCudaErrors(hipPeekAtLastError());
        printf("Done trial %i\n", i);
    }
    sdkStopTimer(&hTimer);

    printf("%f Grays/s\n", (float)TRIALS * samples * 1e-9 * 1e+3 / sdkGetTimerValue(&hTimer));

    //float3* rays = new float3[samples];
    //CUDA_CALL(hipMemcpy(rays, devRaysDirection, sizeof(float3) * samples, hipMemcpyDeviceToHost));
    //delete rays;
    
    hipFree(devRaysDirection);

    hipFree(states);

    return EXIT_SUCCESS;
}